#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stddef.h>

// function to add the elements of two arrays
__global__ void add(int n, float *x, float *y) {
  int iSliceSize = n/(blockDim.x*gridDim.x);
  int iInit      = threadIdx.x*iSliceSize + blockIdx.x*iSliceSize*blockDim.x;
  printf("Thread  %d, block %d, silice size %d, init %d, end %d\n",
    threadIdx.x, blockIdx.x, iSliceSize, iInit, iInit + iSliceSize - 1);
  for (int iIndex = iInit; iIndex < iInit + iSliceSize; iInit++) {
    y[iIndex] = x[iIndex] + y[iIndex];
  }
}

int main(void) {
  int N = 1<<20; // 1M elements
  printf("N is %d\n", N);

  float *x = new float[N];
  float *y = new float[N];
  //float *x;
  //float *y;
  //cudaMallocManaged(&x, N*sizeof(float));
  //cudaMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  
  int iBlocks  = 10000;
  int iThreads = 100;
  printf("Will run GPU concurrency with %d blocks and %d threads per block\n",
    iBlocks, iThreads);
  printf("Just about to execute GPU\n");
  // Run kernel on 1M elements on the CPU
  add<<<iBlocks, iThreads>>>(N, x, y);
  hipDeviceSynchronize();
  printf("Out of GPU region\n");

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  delete [] x;
  delete [] y;
  //cudaFree(x);
  //cudaFree(y);

  return 0;
}
