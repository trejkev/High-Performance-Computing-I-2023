#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stddef.h>

// function to add the elements of two arrays
__global__ void add(int n, float *x, float *y) {
  int iSliceSize = n/(blockDim.x*gridDim.x);
  int iInit      = threadIdx.x*iSliceSize + blockIdx.x*iSliceSize*blockDim.x;
  int iEnd       = iInit + iSliceSize;
  
  // Adjust iEnd if this is the last block and thread
  if (blockIdx.x == gridDim.x - 1 && threadIdx.x == blockDim.x - 1) {
    iEnd = n;
  }
  
  // add the input data
  for (int iIndex = iInit; iIndex < iEnd; ++iIndex) {
    y[iIndex] += x[iIndex];
  }
}

int main(void) {
  int N = 1<<20; // 1M elements
  printf("N is %d\n", N);

  float *fXHost = (float*)calloc(N, sizeof(float));
  float *fYHost = (float*)calloc(N, sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    fXHost[i] = 1.0f;
    fYHost[i] = 2.0f;
  }
  
  // Allocate memory for device
  float *fXDevice;
  float *fYDevice;
  hipMalloc((void**)&fXDevice, N*sizeof(float));
  hipMalloc((void**)&fYDevice, N*sizeof(float));
  hipMemcpy(fXDevice, fXHost, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(fYDevice, fYHost, N*sizeof(float), hipMemcpyHostToDevice);

  int iBlocks  = 10000;
  int iThreads = 100;
  printf("Will run GPU concurrency with %d blocks and %d threads per block\n", iBlocks, iThreads);
  printf("Just about to execute GPU\n");
  // Run kernel on 1M elements on the CPU
  add<<<iBlocks, iThreads>>>(N, fXDevice, fYDevice);
  hipDeviceSynchronize();
  printf("Out of GPU region\n");

  hipMemcpy(fYHost, fYDevice, N*sizeof(float), hipMemcpyDeviceToHost);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(fYHost[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  //delete [] x;
  //delete [] y;
  hipFree(fXDevice);
  hipFree(fYDevice);
  free(fXHost);
  free(fYHost);

  return 0;
}
