
// FIXME using strides

#include <iostream>
#include <cstdio>
#include <stddef.h>
#include <hip/hip_runtime.h>


void init(int *a, int N) {
  for (int i = 0; i < N; ++i) {
    a[i] = i;
  }
}

__global__ void doubleElements(int *a, int N) {
  int iSliceSize = N/(blockDim.x*gridDim.x);
  int iInit      = threadIdx.x*iSliceSize + blockIdx.x*iSliceSize*blockDim.x;
  int iEnd       = iInit + iSliceSize;

  // Adjust iEnd if this is the last block and thread
  if (blockIdx.x == gridDim.x - 1 && threadIdx.x == blockDim.x - 1) {
    iEnd = N;
  }
  
  // Double the input data
  for (int iIndex = iInit; iIndex < iEnd; iIndex++) {
    a[iIndex] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N) {
  for (int i = 0; i < N; ++i) {
    if (a[i] != i*2) {
      printf("Element %d not doubled\n", a[i]);
      return false;
    }
  }
  return true;
}

int main()
{
  /*
   * 'N' is greater than the size of the grid (see below).
   */

  int N = 10000;
  int *b = (int*)calloc(N, sizeof(int)); // Host memory allocation

  init(b, N);

  int *a;
  hipMalloc((void**)&a, N*sizeof(int)); // Device memory pointer
  hipMemcpy(a, b, N * sizeof(int), hipMemcpyHostToDevice); // Copy memory contents from host to device

  /*
   * The size of this grid is 256*32 = 8192.
   */

  size_t threads_per_block = 256;
  size_t number_of_blocks = 32;
  
  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  hipMemcpy(b, a, N * sizeof(int), hipMemcpyDeviceToHost); // Copy memory contents from device to host

  bool areDoubled = checkElementsAreDoubled(b, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
  free(b);
}
